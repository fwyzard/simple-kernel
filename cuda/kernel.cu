#include <cstdio>

#include <hip/hip_runtime.h>


__global__
void kernel(const char* message) {
  printf("%s\n", message);
}
